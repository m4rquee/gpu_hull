#include "../include/gpu_hull/gpu_vector.cuh"
#include <cassert>
#include <hip/hip_runtime.h>

int main() {
    typedef struct {
        float x;
        float y;
    } Point;

    const int size = 10000;
    auto harrayA = new int[size];
    auto harrayB = new Point[size];
    harrayA[0] = harrayA[size - 1] = -1;
    harrayB[0].x = harrayB[size - 1].y = -1;

    hipStream_t streamA;
    hipStreamCreateWithFlags(&streamA, hipStreamNonBlocking);
    hipStream_t streamB;
    hipStreamCreateWithFlags(&streamB, hipStreamNonBlocking);

    GPUVector<int> arrayA(size, streamA);
    arrayA.memset(0);
    GPUVector<Point> arrayB(size, streamB);
    arrayB.memset(0);

    arrayA.memcpy(harrayA);
    arrayB.memcpy(harrayB);

    hipStreamSynchronize(streamA);
    hipStreamSynchronize(streamB);
    assert(harrayA[0] == 0);
    assert(harrayA[size - 1] == 0);
    assert(harrayB[0].x == 0);
    assert(harrayB[size - 1].y == 0);

    arrayA.free();
    arrayB.free();
    hipStreamDestroy(streamA);
    hipStreamDestroy(streamB);
    delete[] harrayA;
    delete[] harrayB;
    return 0;
}
